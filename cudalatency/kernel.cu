#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>

#include "../common/timing.h"
#include "../common/timing.c"

#define CACHELINE_SIZE 64

int default_test_sizes[] = { 2, 4, 8, 16, 24, 32, 48, 64, 96, 128, 192, 256, 384, 512, 600, 768, 1024, 
    1536, 2048, 3072, 4096, 5120, 6144, 8192, 16384, 32768, 65536, 98304, 131072, 196608, 262144, 524288, 1048576 };

hipError_t TestCudaLatency(int size, uint32_t chase_iterations);

__global__ void latencyKernel(int *a, int *count, int *ret)
{
    int current = a[0];
    for (int i = 0; i < *count; i++) {
        // asm("add.s32 %0, %1, %2;" : "=r"(i) : "r"(j), "r"(k));
        current = a[current];
    }

    ret[0] = current;
}

void FillPatternArr(uint32_t* pattern_arr, uint32_t list_size, uint32_t byte_increment) {
    uint32_t increment = byte_increment / sizeof(uint32_t);
    uint32_t element_count = list_size / increment;
    for (int i = 0; i < element_count; i++) {
        pattern_arr[i * increment] = i * increment;
    }

    int iter = element_count;
    while (iter > 1) {
        iter -= 1;
        int j = iter - 1 == 0 ? 0 : rand() % (iter - 1);
        uint32_t tmp = pattern_arr[iter * increment];
        pattern_arr[iter * increment] = pattern_arr[j * increment];
        pattern_arr[j * increment] = tmp;
    }
}

int main(int argc, char *argv[])
{
    int iter_mul = 7;
    if (argc > 1) { iter_mul = atoi(argv[1]); }
    if (argc > 2) {
        if (argv[2][0] == 'l') hipDeviceSetCacheConfig(hipFuncCachePreferL1);
        else if (argv[2][0] == 's') hipDeviceSetCacheConfig(hipFuncCachePreferShared);
        else if (argv[2][0] == 'e') hipDeviceSetCacheConfig(hipFuncCachePreferEqual);
    }

    fprintf(stderr, "Iterations: %d mil\n", iter_mul);
    // warmup
    TestCudaLatency(2, 1e6 * iter_mul);
    for (int size_idx = 0; size_idx < sizeof(default_test_sizes) / sizeof(int); size_idx++) {
        TestCudaLatency(default_test_sizes[size_idx], 1e6 * iter_mul);
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t TestCudaLatency(int sizeKb, uint32_t chase_iterations)
{
    int *dev_a = 0;
    int *dev_count = 0;
    int *dev_result = 0;
    int64_t time_diff_ms;
    float latency;
    hipError_t cudaStatus;

    uint32_t result = 0;
    int list_size = 256 * sizeKb;
    uint32_t* A = (uint32_t*)malloc(sizeof(uint32_t) * list_size);
    FillPatternArr((uint32_t*)A, list_size, CACHELINE_SIZE);

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // allocate memory for pattern array
    cudaStatus = hipMallocManaged((void**)&dev_a, sizeof(uint32_t) * list_size, hipMemAttachGlobal);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocManaged failed!");
        goto Error;
    }

    hipMemAdvise((void*)dev_a, sizeof(uint32_t) * list_size, hipMemAdviseSetReadMostly, 0);

    // copy result to GPU
    cudaStatus = hipMemcpy(dev_a, A, list_size * sizeof(uint32_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_count, sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_count, &chase_iterations, sizeof(uint32_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_result, sizeof(uint32_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    start_timing();
    latencyKernel<<<1, 1>>>(dev_a, dev_count, dev_result);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    time_diff_ms = end_timing();
    latency = 1e6 * (float)time_diff_ms / (float)chase_iterations;
    printf("%d,%f\n", sizeKb, latency);

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(&result, dev_result, sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_result);
    hipFree(dev_a);
    hipFree(dev_count);
    
    return cudaStatus;
}
